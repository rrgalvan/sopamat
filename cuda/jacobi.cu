#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define JACOBI_DEBUG 1
#define TOL 0.0001

__device__ int flag;

__global__ void internal_jacobi_solve(float * a, float * x0, float * b, unsigned int matrixSize) {
	unsigned int i, j;
	float sigma = 0, newValue;

	i = threadIdx.x + blockIdx.x * blockDim.x;

	for (j = 0; j < matrixSize; j++) {
		if (i != j) {
			sigma = sigma + a[i*matrixSize + j] * x0[j];
		}
	}

	newValue = (b[i] - sigma) / a[i*matrixSize + i];

	if (abs(x0[i] - newValue) > TOL) flag = 0;
	x0[i] = newValue;
}
hipError_t cuda_jacobi_solve(float ** a, float * x0, float * b, unsigned int matrixSize, int * iter) {
	unsigned int i, j;
	int  blockSize, minGridSize, gridSize, cpuConvergenceTest, k;
	float *extended_a = 0, *dev_a = 0, *dev_x0 = 0, *dev_b = 0, *dev_old = 0;

	hipError_t cudaStatus;

	if (JACOBI_DEBUG) printf("Hello GPU0\n");
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	if (JACOBI_DEBUG) printf("Hello CPU\n");


	if (JACOBI_DEBUG) printf("Generating a vector with NxN size with a values\n");
	extended_a = (float*)malloc(matrixSize*matrixSize*sizeof(float));
	for (i = 0; i < matrixSize; i++) {
		for (j = 0; j < matrixSize; j++) {
			extended_a[i*matrixSize + j] = a[i][j];
		}
	}
	if (JACOBI_DEBUG) printf("Generated a vector with NxN size with a values\n");


	if (JACOBI_DEBUG) printf("Allocating memory in GPU\n");

	cudaStatus = hipMalloc((void**)&dev_a, matrixSize*matrixSize*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_x0, matrixSize*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, matrixSize*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_old, matrixSize*sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	if (JACOBI_DEBUG) printf("Allocated memory in GPU\n");

	if (JACOBI_DEBUG) printf("Copying memory in GPU\n");

	cudaStatus = hipMemcpy(dev_a, extended_a, matrixSize*matrixSize* sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_x0, x0, matrixSize* sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_old, x0, matrixSize* sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, matrixSize* sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cpuConvergenceTest = 0;
	k = 0;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, internal_jacobi_solve, 0, matrixSize);
	gridSize = (matrixSize + blockSize - 1) / blockSize;

	for (i = 0; i < 100 && !cpuConvergenceTest; i++) {
		cpuConvergenceTest = 1;
		hipMemcpyToSymbol(HIP_SYMBOL(flag), &cpuConvergenceTest, sizeof(int));


		internal_jacobi_solve << <gridSize, blockSize >> >(dev_a, dev_x0, dev_b, matrixSize);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching jacobi!\n", cudaStatus);
			goto Error;
		}

		hipMemcpyFromSymbol(&cpuConvergenceTest, HIP_SYMBOL(flag), sizeof(int));

		k++;
	}


	*iter = k;
	cudaStatus = hipMemcpy(x0, dev_x0, matrixSize* sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	free(extended_a);
	hipFree(dev_a);
	hipFree(dev_x0);
	hipFree(dev_b);
	return cudaStatus;
}

int main()
{
	unsigned int matrixSize = 1000, i, j;
	int iter;
	float *x0, *b, **test;
	// Allocate memory for CPU.
	test = (float**)malloc(sizeof(float*)*matrixSize);
	b = (float*)malloc(sizeof(float)*matrixSize);
	x0 = (float*)malloc(sizeof(float)*matrixSize);
	for (i = 0; i < matrixSize; i++) {
		test[i] = (float*)calloc(matrixSize, sizeof(float));

		// ONLY FOR TESTING PURPOSE.
		test[i][i] = 2;
		test[i][1] = 2*i;
		b[i] = 3;
		x0[i] = 0;
	}

	cuda_jacobi_solve(test, x0, b, matrixSize, &iter);

	// You can enable printing with nvcc -D JACOBI_PRINT_SOLUTION
	#ifdef JACOBI_PRINT_SOLUTION
	printf("x0=(");
	for (i = 0; i < matrixSize; i++) {
		printf("%f,", x0[i]);
	}
	printf(")");
	#endif

	system("pause");

	return 0;
}
