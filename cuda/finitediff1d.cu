#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

typedef enum {
	CONVERGENCE, CUDAFAIL, NOCONVERGENCE
} poison_status;


typedef struct t_problem_poison1d  {
	float tol, alpha, beta, a, b;
	unsigned int max_iter, n;

} problem_poison1d;

typedef struct t_solution_poison1d  {
	poison_status status;
	float * u;
} solution_poison1d;


__device__ int flag;


__device__ float f(float x) {
	return -12*x*x;
}

__global__ void solve_poison1d_kernel(const float h, float * u, const float tol, float a, float b, int n) {
	unsigned int i;
	float newValue, xi;

	i = threadIdx.x + blockIdx.x * blockDim.x+ 1;
	xi = a + i*(b - a) / (n + 1);

	newValue = 0.5*(u[i - 1] + u[i + 1] + h*h*(f(xi)));

	// Convergence test
	if (abs(u[i] - newValue) > tol) flag = 0;
	u[i] = newValue;
}

solution_poison1d solve_poison1d(problem_poison1d p, int * iter) {

	int  blockSize, minGridSize, gridSize, i, k, cpuConvergenceTest;
	float *dev_u, h;
	hipError_t cudaStatus;

	solution_poison1d sp;
	h = (-p.a + p.b) / (p.n + 1);


	printf("h=%f\n", h);
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		sp.status = CUDAFAIL;
		return sp;
	}

	sp.u = (float*)malloc(sizeof(float)*(p.n + 1));
	cudaStatus = hipMalloc((void**)&dev_u, sizeof(float)*(p.n + 1));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		sp.status = CUDAFAIL;
		return sp;
	}
	cudaStatus = hipMalloc((void**)&flag, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		sp.status = CUDAFAIL;
		return sp;
	}
	for (i = 0; i < (p.n + 2); i++) sp.u[i] = p.alpha + i*(p.beta - p.alpha) / (p.n + 1);

	cudaStatus = hipMemcpy(dev_u, sp.u, sizeof(float)*(p.n + 1), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		sp.status = CUDAFAIL;
		return sp;
	}
	cpuConvergenceTest = 0;
	k = 0;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, solve_poison1d_kernel, 0, p.n-1);
	gridSize = (p.n-1 + blockSize - 1) / blockSize;
	for (i = 0; i < p.max_iter && !cpuConvergenceTest; i++) {
		cpuConvergenceTest = 1;

		hipMemcpyToSymbol(HIP_SYMBOL(flag), &cpuConvergenceTest, sizeof(int));

		solve_poison1d_kernel << <gridSize, blockSize >> >(h, dev_u, p.tol, p.a, p.b, p.n);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			sp.status = CUDAFAIL;
			return sp;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching solve_poison1_kernel!\n", cudaStatus);

			sp.status = CUDAFAIL;
			return sp;
		}
		hipMemcpyFromSymbol(&cpuConvergenceTest, HIP_SYMBOL(flag), sizeof(int));

		k++;
	}
	hipMemcpy(sp.u, dev_u, sizeof(float)*(p.n + 1), hipMemcpyDeviceToHost);

	*iter = k;
	if (k == p.max_iter) {
		sp.status = NOCONVERGENCE;
		return sp;
	}

	sp.status = CONVERGENCE;
	return sp;
}


float evaluate_solution(solution_poison1d s, float x, float a, float b, float n) {
	int i;
	i = (x - a)*(n + 1) / (b - a);
	return s.u[i];
}
int main()
{
	int iter = 0, i;
	problem_poison1d p;

	p.tol = powf(10, -5);
	p.max_iter = 10000;
	p.alpha = 0;
	p.beta = 1;
	p.a = 0;
	p.b = 1;
	p.n = 100;

	solution_poison1d s = solve_poison1d(p, &iter);

	switch (s.status) {
		case CONVERGENCE:
			printf("%d\n", iter);

			for (i = 0; i < p.n + 2; i++) {
				printf("u_%d=%f\n", i, s.u[i]);
			}
			break;
		case NOCONVERGENCE:
			printf("Method doesn't converge");
			break;
		case CUDAFAIL:
			printf("Something is wrong with CUDA");
			break;
	}

	#ifdef _DEBUG
	system("pause");
	#endif
    return 0;
}
